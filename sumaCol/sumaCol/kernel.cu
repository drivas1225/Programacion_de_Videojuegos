
#ifdef _INTELLISENSE_
void __syncthreads();
#endif


#include <stdio.h>
#include <iostream>

#include "hip/hip_runtime.h"


#include <hip/device_functions.h>

#define DIMBLOCKX 32


template<class T>
void print_vector(T* v, unsigned s) {
    for (unsigned i = 0; i < s; i++) {
        std::cout << v[i] << " ";
    }
    std::cout << "\n";
}


__global__ void SumaColMatrizKernel(int M, float* Md, float* Nd) {
    __shared__ float Nds[DIMBLOCKX];
    float Pvalue = 0;
    int columna = blockIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Pvalue = Pvalue + Md[posIni + k];
    }
    Nds[threadIdx.x] = Pvalue; 
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int i = 1; i < blockDim.x; ++i) {
            Nds[0] = Nds[0] + Nds[i];
        }
        Nd[blockIdx.x] = Nds[0];
    }
}


void SumaColMatriz(int M, int N, int* Mh, int* Nh) {
    int size = M * N * sizeof(float), size2 = N * sizeof(float);
    float* Md, * Nd;
    // Allocate en device
    hipMalloc(&Md, size);
    hipMalloc(&Nd, size2);
    // Inicializo matrices en el device
    // Inicializo matrices en el device
    hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
    hipMemset(Nd, 0, size2);

    // Invocar el kernel que suma en GPU

    // configuraci�n de la ejecuci�n
    int chunk = 32;
    dim3 tamGrid(N, 1); //Grid dimensi�n
    dim3 tamBlock(M / chunk,1, 1); //Block dimensi�n
    // lanzamiento del kernel
    SumaColMatrizKernel <<<tamGrid, tamBlock >>> (M, Md, Nd);

    // Traer resultado;
    hipMemcpy(Nh, Nd, size2, hipMemcpyDeviceToHost);
    // Free matrices en device
}

int main()
{
    int M = 32;
    int N = 64;

    int* Mh = new int[M * N];
    int* Rh = new int[N];

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            Mh[(i * N) + j] = j;
        }
    }


    SumaColMatriz(M, N, Mh, Rh);
    print_vector(Rh, N);

    delete[] Mh;
    delete[] Rh;



    return 0;
}

