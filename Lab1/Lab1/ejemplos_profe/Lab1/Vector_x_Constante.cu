#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

//DEVICE

__global__ void kernelVector_x_constant( float* arr, int n, int k )
{
	//Obtengo el indice del hilo fisico
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//Mientras el hilo sea valido para la operaci�n
	if( idx<n )
	{
		//Multiplico el elemento por la constante
		arr[ idx ] = arr[ idx ] * k;
	}
}

//HOST
void main()
{
	int size = 1000000;
	//Separo memoria en la RAM del HOST
	float* arr = new float[size];
	float* arr_DEVICE = NULL;

	//Inicializo el arreglo en el HOST
	for( int index = 0; index<size ; index++ )
	{
		arr[index] = index;
	}

	//Separo memoria en la RAM del DEVICE ( la misma cantidad de bytes que en el HOST )
	hipMalloc((void**)&arr_DEVICE, size * sizeof(float));

	//Copio el bloque de memoria del HOST al DEVICE
	hipMemcpy( arr_DEVICE, arr, size * sizeof(float), hipMemcpyHostToDevice);
	
	///////////////////////// EJECUTO EL KERNEL DE CUDA ////////////////////////////
	//////// 512 Hilos
	//////// ceil(1000000/512) Bloques
	kernelVector_x_constant<<< ceil(size/512.0), 512 >>>( arr_DEVICE, size, 65 );
	//Fuerzo una llamada Sincrona
	hipDeviceSynchronize();

	//Copio mis datos ya procesados a la RAM del HOST 
	hipMemcpy( arr, arr_DEVICE, size * sizeof(float), hipMemcpyDeviceToHost);

	//Con una impresi�n de los primeros 100 visualizo el resultado
	for( int index = 0; index<100 ; index++ )
	{
		cout<<arr[index]<<endl;
	}
	
	//Libero memoria en la RAM del DEVICE
	hipFree( arr_DEVICE );

	//Libero memoria en la RAM del HOST
	delete[] arr;
	cin.get();
}
