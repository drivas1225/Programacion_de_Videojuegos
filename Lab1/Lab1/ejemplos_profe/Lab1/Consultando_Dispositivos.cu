#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

void main()
{
	hipDeviceProp_t prop;	

	int count = 0;

	hipGetDeviceCount( &count );

	for( int index=0; index<count ;index++ )
	{
		hipGetDeviceProperties(&prop, index);

		cout<<prop.name<<endl;
		cout<<prop.minor<<" - "<<prop.major<<endl;
		cout<<prop.clockRate<<endl;

	}	


	cin.get();
}
