#include "hip/hip_runtime.h"

	
#include <stdio.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


using namespace std;

//DEVICE

__global__ void kernelVector_x_constant(float* arr, int n, int k)
{
	//Obtengo el indice del hilo fisico
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//Mientras el hilo sea valido para la operacion
	if (idx < n)
	{
		//Multiplico el elemento por la constante
		arr[idx] = arr[idx] * k;
	}
}

__global__ void KernelVector_add_constant(float* arr, float* res , int n, int c) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n)
	{
		res[idx] = arr[idx] + c;
	}
}

__global__ void KernelVector_sub_constant(float* arr, float* res, int n, int c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n)
	{
		res[idx] = arr[idx] - c;
	}
}

__global__ void KernelVector_add_vector(float *arr1 , float *arr2 , float *res  , int n )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n)
	{
		res[idx] = arr1[idx] + arr2[idx];
	}
}

__global__ void KernelVector_sub_vector(float* arr1, float* arr2, float* res, int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n)
	{
		res[idx] = arr1[idx] - arr2[idx];
	}
}

//HOST
int main()
{
	int size = 1000000;
	//Separo memoria en la RAM del HOST
	float* arrA;
	float* arrB;
	float e = 5;
	float* res;

	hipMallocManaged(&arrA, size * sizeof(float));
	hipMallocManaged(&arrB, size * sizeof(float));
	hipMallocManaged(&res , size * sizeof(float));

	for (int i = 0; i < size; i++) {
		arrA[i] = i;
		arrB[i] = i;
		res[i] = 0;
	}


	///////////////////////// EJECUTO EL KERNEL DE CUDA ////////////////////////////
	//////// 512 Hilos
	//////// ceil(1000000/512) Bloques
	//kernelVector_x_constant <<< ceil(size / 512.0), 512 >>> (arr_DEVICE, size, 65);
	
	//// tarea ////
	KernelVector_add_constant <<< ceil(size / 512.0), 512 >>> (arrA,res,size,e);
	//KernelVector_sub_constant <<< ceil(size / 512.0), 512 >>> (arrA, res, size, e);
	
	//KernelVector_add_vector <<< ceil(size / 512.0), 512 >>> (arrA, arrB, res, size);
	//KernelVector_sub_vector <<< ceil(size / 512.0), 512 >>> (arrA, arrB, res, size);


	//Fuerzo una llamada Sincrona
	hipDeviceSynchronize();

	for (int index = 0; index < size; index++)
	{
		cout << res[index] << " ; ";
	}



	hipFree(arrA);
	hipFree(arrB);
	hipFree(res);

	return 0;
}